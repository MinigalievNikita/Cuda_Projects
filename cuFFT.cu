#include <iostream> 
#include <hipfft/hipfft.h> 
#include <hip/hip_runtime.h> 
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#pragma comment(lib, "D:/Cuda_Juliya/lib/glut64.lib")
// "D:/Cuda_Juliya/bin/glut64.dll"
#include "D:/Cuda_Juliya/common/book.h"
#include "D:/Cuda_Juliya/common/cpu_bitmap.h"
#include <stdio.h>
#include <time.h>

#define N 32768 // Size of the FFT, Must be a power of 2 


int main() 
{ 
	hipfftHandle plan;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float sumtime = 0;
	for (int j = 0; j < 10; ++j)
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		
		hipfftComplex h_input[N]; // Input array (complex numbers) 
		hipfftComplex h_output[N]; // Output array (complex numbers) 

		for (int i = 0; i < N; ++i)
		{
			h_input[i].x = i; // Real part 
			h_input[i].y = 0; // Imaginary part 
		}

		hipfftComplex* d_input, * d_output;
		hipMalloc((void**)&d_input, sizeof(hipfftComplex) * N);
		hipMalloc((void**)&d_output, sizeof(hipfftComplex) * N);
    
		hipMemcpy(d_input, h_input, sizeof(hipfftComplex) * N, hipMemcpyHostToDevice);
    
		hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); // HIPFFT_C2C for complex-to-complex transform 

		hipEventRecord(start, 0);

		hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD);
		
    hipEventRecord(stop, 0);
		
    hipDeviceSynchronize();
		
    hipMemcpy(h_output, d_output, sizeof(hipfftComplex) * N, hipMemcpyDeviceToHost);

		hipEventSynchronize(stop);
		
    float elapsedTime;
		
    hipEventElapsedTime(&elapsedTime, start, stop); 
		
    sumtime += elapsedTime;
		
		hipEventDestroy(start);
		hipEventDestroy(stop);
		hipfftDestroy(plan);
		hipFree(d_input);
		hipFree(d_output);
		std::cout << "FFT Output:" << std::endl;
		for (int i = 0; i < 8; ++i)
		{
			std::cout <<  " (" << h_output[i].x << ", " << h_output[i].y << ")" << std::endl;
		}
	}
	printf("Time taken: %f s\n", sumtime/10);
	return 0; 
} 
